#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_boundaries.cu
 \brief Implementation of boundary condition functions with OpenMP threading
*/

#include <math.h>
#include <omp.h>

extern "C" {
#include "boundaries.h"
}

#include "cuda_kernels.cuh"

__constant__ fp_t d_bc[2][2];

void set_boundaries(fp_t bc[2][2])
{
	/* Change these values to your liking: */
	fp_t clo = 0.0, chi = 1.0;

	bc[0][0] = clo; /* bottom boundary */
	bc[0][1] = clo; /* top boundary */
	bc[1][0] = chi; /* left boundary */
	bc[1][1] = chi; /* right boundary */
}

void apply_initial_conditions(fp_t** conc, const int nx, const int ny, const int nm, fp_t bc[2][2])
{
	#pragma omp parallel
	{
		#pragma omp for collapse(2)
		for (int j = 0; j < ny; j++)
			for (int i = 0; i < nx; i++)
				conc[j][i] = bc[0][0];

		#pragma omp for collapse(2)
		for (int j = 0; j < ny/2; j++)
			for (int i = 0; i < 1+nm/2; i++)
				conc[j][i] = bc[1][0]; /* left half-wall */

		#pragma omp for collapse(2)
		for (int j = ny/2; j < ny; j++)
			for (int i = nx-1-nm/2; i < nx; i++)
				conc[j][i] = bc[1][1]; /* right half-wall */
	}
}

__global__ void boundary_kernel(fp_t* d_conc,
                                const int nx,
                                const int ny,
                                const int nm)
{
	/* determine indices on which to operate */
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int row = blockDim.y * blockIdx.y + ty;
	const int col = blockDim.x * blockIdx.x + tx;

	/* apply fixed boundary values: sequence does not matter */

	if (row < ny/2 && col < 1+nm/2) {
		d_conc[row * nx + col] = d_bc[1][0]; /* left value */
	}

	if (row >= ny/2 && row < ny && col >= nx-1-nm/2 && col < nx) {
		d_conc[row * nx + col] = d_bc[1][1]; /* right value */
	}

	/* wait for all threads to finish writing */
	__syncthreads();

	/* apply no-flux boundary conditions: inside to out, sequence matters */

	for (int offset = 0; offset < nm/2; offset++) {
		const int ilo = nm/2 - offset;
		const int ihi = nx - 1 - nm/2 + offset;
		const int jlo = nm/2 - offset;
		const int jhi = ny - 1 - nm/2 + offset;

		if (ilo-1 == col && row < ny) {
			d_conc[row * nx + ilo-1] = d_conc[row * nx + ilo]; /* left condition */
		}
		if (ihi+1 == col && row < ny) {
			d_conc[row * nx + ihi+1] = d_conc[row * nx + ihi]; /* right condition */
		}
		if (jlo-1 == row && col < nx) {
			d_conc[(jlo-1) * nx + col] = d_conc[jlo * nx + col]; /* bottom condition */
		}
		if (jhi+1 == row && col < nx) {
			d_conc[(jhi+1) * nx + col] = d_conc[jhi * nx + col]; /* top condition */
		}

		__syncthreads();
	}
}
