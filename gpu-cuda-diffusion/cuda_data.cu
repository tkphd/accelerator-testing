/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_data.cu
 \brief Implementation of functions to create and destroy CudaData struct
*/

extern "C" {
#include "cuda_data.h"
}

#include "cuda_kernels.cuh"

void init_cuda(fp_t** conc_old, fp_t** mask_lap, fp_t bc[2][2],
               const int nx, const int ny, const int nm, struct CudaData* dev)
{
	/* allocate memory on device */
	hipMalloc((void **) &(dev->conc_old), nx * ny * sizeof(fp_t));
	hipMalloc((void **) &(dev->conc_lap), nx * ny * sizeof(fp_t));
	hipMalloc((void **) &(dev->conc_new), nx * ny * sizeof(fp_t));

	/* transfer mask and boundary conditions to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_bc), bc[0], 2 * 2 * sizeof(fp_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer data from host in to GPU */
	hipMemcpy(dev->conc_old, conc_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
}

void free_cuda(struct CudaData* dev)
{
	/* free memory on device */
	hipFree(dev->conc_old);
	hipFree(dev->conc_lap);
	hipFree(dev->conc_new);
}
